#include "hip/hip_runtime.h"
﻿#include "MogiAccelerator.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <chrono>
#include <iostream>

#include "src/Kernels.h"


namespace_accelerator_start

void CudaDealloc(float* devicePtr)
{
    hipFree(devicePtr);
}

float* CudaAlloc(size_t count)
{
    float* devicePtr = nullptr;
    size_t size = count * sizeof(float);
    hipMalloc(&devicePtr, size);
    return devicePtr;
}

void CudaCopyHostToDevice(float* device, const float* host, size_t size)
{
    hipMemcpy(device, host, size * sizeof(float), hipMemcpyHostToDevice);
}

void CudaCopyDeviceToHost(float* host, const float* device, size_t size)
{
    hipMemcpy(host, device, size * sizeof(float), hipMemcpyDeviceToHost);
}

void CudaCopyDeviceToDevice(float* deviceDst, const float* deviceSrc, size_t size)
{
    hipMemcpy(deviceDst, deviceSrc, size * sizeof(float), hipMemcpyDeviceToDevice);
}

void CudaCopyHostToDevice(Tensor* device, Tensor* host)
{
    hipMemcpy(device->GetData(), host->GetData(), host->GetSize() * sizeof(float), hipMemcpyHostToDevice);
}

void CudaCopyDeviceToHost(Tensor* host, Tensor* device)
{
    hipMemcpy(host->GetData(), device->GetData(), device->GetSize() * sizeof(float), hipMemcpyDeviceToHost);
}

void CudaCopyDeviceToDevice(Tensor* deviceDst, const Tensor* deviceSrc)
{
    int blockSize = std::min(size_t(256), deviceSrc->GetSize());
    int numBlocks = (deviceSrc->GetSize() + blockSize - 1) / blockSize;
    CopyKernel << <numBlocks, blockSize >> > (deviceDst->GetData(), deviceSrc->GetData(), deviceSrc->GetSize());
    hipDeviceSynchronize();
}

void CudaMemSet(Tensor* device, size_t size, float value)
{
    int blockSize = std::min(size_t(256), size);
    int numBlocks = (size + blockSize - 1) / blockSize;
    CopyKernel<<<numBlocks, blockSize>>>(device->GetData(), value, size);
    hipDeviceSynchronize();
}

float* ToDevicePtr(const Tensor2D& tensor, bool copy=true)
{
    float* dPtr;
    size_t size = tensor.GetSize() * sizeof(float);

    hipMalloc(&dPtr, size);
    if (copy)
    {
        hipMemcpy(dPtr, tensor.GetData(), size, hipMemcpyHostToDevice);
    }

    return dPtr;
}

void CopyToHost(Tensor2D& dest, float* deviceSrource)
{
    hipMemcpy(dest.GetData(), deviceSrource, dest.GetSize() * sizeof(float), hipMemcpyDeviceToHost);
}

Tensor2D CudaRandom2D(size_t rows, size_t cols, float min, float max)
{
    Tensor2D res(rows, cols, 0.0f, true);
    int blockSize = std::min(size_t(256), res.GetSize());
    int numBlocks = (res.GetSize() + blockSize - 1) / blockSize;
    RandomizeKernel<<<numBlocks, blockSize>>>(res.GetData(), res.GetSize(), min, max);
    return res;
}

void CudaAdd(Tensor* device, float value)
{
    int blockSize = std::min(size_t(256), device->GetSize());
    int numBlocks = (device->GetSize() + blockSize - 1) / blockSize;
    AddKernel << <numBlocks, blockSize >> > (device->GetData(), device->GetSize(), value);
}

void CudaSub(Tensor* device, float value)
{
    int blockSize = std::min(size_t(256), device->GetSize());
    int numBlocks = (device->GetSize() + blockSize - 1) / blockSize;
    SubKernel << <numBlocks, blockSize >> > (device->GetData(), device->GetSize(), value);
}

void CudaMult(Tensor* device, float value)
{
    int blockSize = std::min(size_t(256), device->GetSize());
    int numBlocks = (device->GetSize() + blockSize - 1) / blockSize;
    MultKernel << <numBlocks, blockSize >> > (device->GetData(), device->GetSize(), value);
}

void CudaDiv(Tensor* device, float value)
{
    int blockSize = std::min(size_t(256), device->GetSize());
    int numBlocks = (device->GetSize() + blockSize - 1) / blockSize;
    DivKernel << <numBlocks, blockSize >> > (device->GetData(), device->GetSize(), value);
}

void CudaAdd(Tensor* device, const Tensor* other)
{
    int blockSize = std::min(size_t(256), device->GetSize());
    int numBlocks = (device->GetSize() + blockSize - 1) / blockSize;
    AddKernel << <numBlocks, blockSize >> > (device->GetData(), other->GetData(), device->GetSize());
}

void CudaSub(Tensor* device, const Tensor* other)
{
    int blockSize = std::min(size_t(256), device->GetSize());
    int numBlocks = (device->GetSize() + blockSize - 1) / blockSize;
    SubKernel << <numBlocks, blockSize >> > (device->GetData(), other->GetData(), device->GetSize());
}

void CudaMult(Tensor* device, const Tensor* other)
{
    int blockSize = std::min(size_t(256), device->GetSize());
    int numBlocks = (device->GetSize() + blockSize - 1) / blockSize;
    MultKernel << <numBlocks, blockSize >> > (device->GetData(), other->GetData(), device->GetSize());
}

void CudaDiv(Tensor* device, const Tensor* other)
{
    int blockSize = std::min(size_t(256), device->GetSize());
    int numBlocks = (device->GetSize() + blockSize - 1) / blockSize;
    DivKernel << <numBlocks, blockSize >> > (device->GetData(), other->GetData(), device->GetSize());
}

void CudaSigmoid(Tensor* device)
{
    int blockSize = std::min(size_t(256), device->GetSize());
    int numBlocks = (device->GetSize() + blockSize - 1) / blockSize;
    SigmoidKernel << <numBlocks, blockSize >> > (device->GetData(), device->GetSize());
}

void CudaDiffSigmoid(Tensor* device)
{
    int blockSize = std::min(size_t(256), device->GetSize());
    int numBlocks = (device->GetSize() + blockSize - 1) / blockSize;
    DiffSigmoidKernel << <numBlocks, blockSize >> > (device->GetData(), device->GetSize());
}

void CudaRelU(Tensor* device, float alpha)
{
    int blockSize = std::min(size_t(256), device->GetSize());
    int numBlocks = (device->GetSize() + blockSize - 1) / blockSize;
    RelUKernel << <numBlocks, blockSize >> > (device->GetData(), device->GetSize(), alpha);
}

void CudaDiffRelU(Tensor* device, float alpha)
{
    int blockSize = std::min(size_t(256), device->GetSize());
    int numBlocks = (device->GetSize() + blockSize - 1) / blockSize;
    DiffRelUKernel << <numBlocks, blockSize >> > (device->GetData(), device->GetSize(), alpha);
}

void CudaAdamOptimization(Tensor* params, Tensor* gradients, Tensor* firstMoments, Tensor* secondMoments, float b1, float b2, float ep, size_t timeStep, float learningRate)
{
    int blockSize = std::min(size_t(256), params->GetSize());
    int numBlocks = (params->GetSize() + blockSize - 1) / blockSize;
    AdamOptimizationKernel<<<numBlocks, blockSize>>>(
        params->GetData(), gradients->GetData(), firstMoments->GetData(), secondMoments->GetData(), 
        params->GetSize(), b1, b2, ep, timeStep, learningRate);
}

Tensor2D CudaCorrectedMoments(Tensor* moments, float b, float timeStep)
{
    Tensor2D res(moments->GetSize(), 1, (const float*)moments->GetData(), true);
    int blockSize = std::min(size_t(256), moments->GetSize());
    int numBlocks = (moments->GetSize() + blockSize - 1) / blockSize;
    CorrectedMomentsKernel<<<numBlocks, blockSize>>>(res.GetData(), res.GetSize(), b, timeStep);
    return res;
}

Tensor2D CudaCorrectedGradient(Tensor* firstMoments, Tensor* secondMoments, float learningRate, float epsilon)
{
    Tensor2D res(firstMoments->GetSize(), 1, 0.0f, true);
    int blockSize = std::min(size_t(256), firstMoments->GetSize());
    int numBlocks = (firstMoments->GetSize() + blockSize - 1) / blockSize;
    CorrectedGradientKernel<<<numBlocks, blockSize>>>(res.GetData(), firstMoments->GetData(), secondMoments->GetData(), res.GetSize(), learningRate, epsilon);
    return res;
}

Tensor2D MatrixMultCUDA(const Tensor2D& left, const Tensor2D& right)
{
    if (left.GetCols() != right.GetRows())
    {
        throw -1;
    }

    Tensor2D res(left.GetRows(), right.GetCols(), 0.0f, true);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((right.GetCols() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (left.GetRows()  + threadsPerBlock.y - 1) / threadsPerBlock.y);
    MatrixMultKernel<< <blocksPerGrid, threadsPerBlock >> > (left.GetData(), right.GetData(), res.GetData(), left.GetRows(), left.GetCols(), right.GetCols());

    return res;
}

Tensor2D MatrixMultRightTransposeCUDA(const Tensor2D& left, const Tensor2D& right)
{
    if (left.GetCols() != right.GetCols())
    {
        throw -1;
    }

    Tensor2D res(left.GetRows(), right.GetRows(), 0.0f, true);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((right.GetRows() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (left.GetRows() + threadsPerBlock.y - 1) / threadsPerBlock.y);

    MatrixMultRightTranposeKernel << <blocksPerGrid, threadsPerBlock >> > (left.GetData(), right.GetData(), res.GetData(), left.GetRows(), left.GetCols(), right.GetRows());

    return res;
}

Tensor2D MatrixMultLeftTransposeCUDA(const Tensor2D& left, const Tensor2D& right)
{
    if (left.GetRows() != right.GetRows())
    {
        throw - 1;
    }

    Tensor2D res(left.GetCols(), right.GetCols(), 0.0f, true);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((right.GetCols() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (left.GetCols() + threadsPerBlock.y - 1) / threadsPerBlock.y);

    MatrixMultLeftTranposeKernel<< <blocksPerGrid, threadsPerBlock >> > (left.GetData(), right.GetData(), res.GetData(), left.GetRows(), left.GetCols(), right.GetCols());

    return res;
}

void CudaConvolution(Tensor2D* output, const Tensor3D* input, const Tensor3D* kernel, size_t stride, size_t padding)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(
        (output->GetCols() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (output->GetRows() + threadsPerBlock.y - 1) / threadsPerBlock.y);
    ConvolutionKernel<<<blocksPerGrid, threadsPerBlock>>>(
        input->GetData(), kernel->GetData(), output->GetData(), 
        input->GetRows(), input->GetCols(), input->GetDepth(),
        kernel->GetRows(), kernel->GetCols(), kernel->GetDepth(),
        output->GetRows(), output->GetCols(),
        stride, padding);
}

void CudaConvolution(Tensor3D* output, const Tensor3D* input, const Tensor2D* kernel, size_t stride, size_t padding)
{
    dim3 threadsPerBlock(16, 16, 4);
    dim3 blocksPerGrid(
        (output->GetCols() + threadsPerBlock.x - 1) / threadsPerBlock.x, 
        (output->GetRows() + threadsPerBlock.y - 1) / threadsPerBlock.y,
        (output->GetDepth() + threadsPerBlock.z - 1) / threadsPerBlock.z);
    Convolution3DKernel<<<blocksPerGrid, threadsPerBlock>>>(
        input->GetData(), kernel->GetData(), output->GetData(),
        input->GetRows(), input->GetCols(), input->GetDepth(),
        kernel->GetRows(), kernel->GetCols(),
        output->GetRows(), output->GetCols(), output->GetDepth(),
        stride, padding);
}

void CudaConvolutionKernelFlip(Tensor3D* output, const Tensor2D* input, const Tensor3D* kernel, size_t stride, size_t padding)
{
    dim3 threadsPerBlock(16, 16, 4);
    dim3 blocksPerGrid(
        (output->GetCols() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (output->GetRows() + threadsPerBlock.y - 1) / threadsPerBlock.y,
        (output->GetDepth() + threadsPerBlock.z - 1) / threadsPerBlock.z);
    Convolution3DBackKernelFlipKernel<<<blocksPerGrid, threadsPerBlock>>>(
        input->GetData(), kernel->GetData(), output->GetData(),
        input->GetRows(), input->GetCols(),
        kernel->GetRows(), kernel->GetCols(), kernel->GetDepth(),
        output->GetRows(), output->GetCols(), output->GetDepth(),
        stride, padding);
}

void CudaMaxPool(Tensor3D* output, const Tensor3D* input, size_t poolHeight, size_t poolWidth)
{
    dim3 threadsPerBlock(16, 16, 4);
    dim3 blocksPerGrid(
        (output->GetCols() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (output->GetRows() + threadsPerBlock.y - 1) / threadsPerBlock.y,
        (output->GetDepth() + threadsPerBlock.z - 1) / threadsPerBlock.z);
    MaxPoolKernel<<<blocksPerGrid, threadsPerBlock>>>(
        input->GetData(), output->GetData(),
        input->GetRows(), input->GetCols(), input->GetDepth(),
        output->GetRows(), output->GetCols(), output->GetDepth(),
        poolHeight, poolWidth);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
    }
}

void CudaDistributeReverseMaxPool(Tensor3D* distributed, const Tensor3D* input, const Tensor3D* output, size_t poolHeight, size_t poolWidth)
{
    dim3 threadsPerBlock(16, 16, 4);
    dim3 blocksPerGrid(
        (output->GetCols() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (output->GetRows() + threadsPerBlock.y - 1) / threadsPerBlock.y,
        (output->GetDepth() + threadsPerBlock.z - 1) / threadsPerBlock.z);
    DistributedReverseMaxPoolKernel << <blocksPerGrid, threadsPerBlock >> > (
        input->GetData(), output->GetData(), distributed->GetData(),
        input->GetRows(), input->GetCols(), input->GetDepth(),
        output->GetRows(), output->GetCols(), output->GetDepth(),
        distributed->GetRows(), distributed->GetCols(), distributed->GetDepth(),
        poolHeight, poolWidth);
}

void CudaNearestUpsample(Tensor3D* output, const Tensor3D* input, size_t upsampleHeight, size_t upsampleWidth)
{
    dim3 threadsPerBlock(16, 16, 4);
    dim3 blocksPerGrid(
        (output->GetCols() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (output->GetRows() + threadsPerBlock.y - 1) / threadsPerBlock.y,
        (output->GetDepth() + threadsPerBlock.z - 1) / threadsPerBlock.z);
    NearestUpsampleKernel<<<blocksPerGrid, threadsPerBlock>>>(
        input->GetData(), output->GetData(),
        input->GetRows(), input->GetCols(), input->GetDepth(),
        output->GetRows(), output->GetCols(), output->GetDepth(),
        upsampleHeight, upsampleWidth);
}

void CudaDistributeReverseNearestUpsample(Tensor3D* distributed, const Tensor3D* output, size_t upsampleHeight, size_t upsampleWidth)
{
    dim3 threadsPerBlock(16, 16, 4);
    dim3 blocksPerGrid(
        (distributed->GetCols() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (distributed->GetRows() + threadsPerBlock.y - 1) / threadsPerBlock.y,
        (distributed->GetDepth() + threadsPerBlock.z - 1) / threadsPerBlock.z);
    DistributeReverseNearestUpsampleKernel<<<blocksPerGrid, threadsPerBlock>>>(
        output->GetData(), distributed->GetData(),
        output->GetRows(), output->GetCols(), output->GetDepth(),
        distributed->GetRows(), distributed->GetCols(), distributed->GetDepth(),
        upsampleHeight, upsampleHeight);
}

void CudaDropOut(Tensor* output, const Tensor* input, float dropoutRate, float retentionProb, Tensor* dropOutMask)
{
    int blockSize = std::min(size_t(256), output->GetSize());
    int numBlocks = (output->GetSize() + blockSize - 1) / blockSize;
    DropOutKernel<<<numBlocks, blockSize>>>(
        input->GetData(), output->GetData(), output->GetSize(), dropoutRate, retentionProb, dropOutMask ? dropOutMask->GetData() : nullptr);
}

Tensor3D CudaCrossEntropyLoss(const Tensor3D* target, const Tensor3D* predictions)
{
    Tensor3D loss = Tensor3D(target->GetRows(), target->GetCols(), target->GetDepth(), 0.0f, true);
    int blockSize = std::min(size_t(256), loss.GetSize());
    int numBlocks = (loss.GetSize() + blockSize - 1) / blockSize;
    CrossEntropyLossKernel<<<numBlocks, blockSize>>>(
        target->GetData(), predictions->GetData(), loss.GetData(), loss.GetSize());
    return loss;
}

namespace_accelerator_end