#include "hip/hip_runtime.h"
#include "Kernels.h"

#include ""
#include "hiprand/hiprand_kernel.h"

#include <math.h>
#include <float.h>


__global__ void CopyKernel(float* tensor, float value, size_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        tensor[idx] = value;
    }
}

__global__ void CopyKernel(float* tensorDst, const float* tensorSrc, size_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        tensorDst[idx] = tensorSrc[idx];
    }
}

__global__ void AddKernel(float* tensor, size_t size, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        tensor[idx] = tensor[idx] + value;
    }
}

__global__ void SubKernel(float* tensor, size_t size, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        tensor[idx] = tensor[idx] - value;
    }
}

__global__ void MultKernel(float* tensor, size_t size, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        tensor[idx] = tensor[idx] * value;
    }
}

__global__ void DivKernel(float* tensor, size_t size, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        tensor[idx] = tensor[idx] / value;
    }
}

__global__ void AddKernel(float* tensor, const float* other, size_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        tensor[idx] = tensor[idx] + other[idx];
    }
}

__global__ void SubKernel(float* tensor, const float* other, size_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        tensor[idx] = tensor[idx] - other[idx];
    }
}

__global__ void MultKernel(float* tensor, const float* other, size_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        tensor[idx] = tensor[idx] * other[idx];
    }
}

__global__ void DivKernel(float* tensor, const float* other, size_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        tensor[idx] = tensor[idx] / other[idx];
    }
}

__global__ void SigmoidKernel(float* tensor, size_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        tensor[idx] = 1.0f / (1.0f + exp(-tensor[idx]));
    }
}

__global__ void DiffSigmoidKernel(float* tensor, size_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        float sigm = 1.0f / (1.0f + exp(-tensor[idx]));
        tensor[idx] = sigm * (1.0f - sigm);
    }
}

__global__ void RelUKernel(float* tensor, size_t size, float alpha)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        tensor[idx] = tensor[idx] > 0 ? tensor[idx] : alpha * tensor[idx];
    }
}

__global__ void DiffRelUKernel(float* tensor, size_t size, float alpha)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        tensor[idx] = tensor[idx] > 0 ? 1.0f : alpha;
    }
}

__global__ void AdamOptimizationKernel(float* params, float* gradients, float* firstMoments, float* secondMoments, size_t size, float b1, float b2, float epsilon, size_t timeStep, float learningRate)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        firstMoments[idx] = b1 * firstMoments[idx] + (1.0f - b1) * gradients[idx];
        secondMoments[idx] = b2 * secondMoments[idx] + (1.0f - b2) * gradients[idx] * gradients[idx];

        float correctedFirstMoment = firstMoments[idx] / (1.0f - pow(b1, timeStep));
        float correctedSecondMoment = secondMoments[idx] / (1.0f - pow(b2, timeStep));

        float correctedGradient = learningRate * correctedFirstMoment / (sqrt(correctedSecondMoment) + epsilon);

        params[idx] = params[idx] - correctedGradient;
    }
}

__global__ void CorrectedMomentsKernel(float* tensor, size_t size, float b, float timeStep)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        tensor[idx] = tensor[idx] / (1.0f - pow(b, timeStep));
    }
}

__global__ void CorrectedGradientKernel(float* res, float* firstMoments, float* secondMoments, size_t size, float learningRate, float epsilon)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        res[idx] = learningRate * firstMoments[idx] / (sqrt(secondMoments[idx]) + epsilon);
    }
}

__global__ void MatrixMultKernel(const float* A, const float* B, float* C, int ARows, int ACols, int BCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < ARows && col < BCols) {
        float sum = 0.0;
        for (int i = 0; i < ACols; ++i) {
            sum += A[row * ACols + i] * B[i * BCols + col];
        }
        C[row * BCols + col] = sum;
    }
}

__global__ void MatrixMultRightTranposeKernel(const float* A, const float* B, float* C, int ARows, int ACols, int BRows)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < ARows && col < BRows) {
        float sum = 0.0f;
        for (int e = 0; e < ACols; ++e) {
            sum += A[row * ACols + e] * B[col * ACols + e]; // Accessing B as if it's transposed
        }
        C[row * BRows + col] = sum;
    }
}

__global__ void MatrixMultLeftTranposeKernel(const float* A, const float* B, float* C, int ARows, int ACols, int BCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < ACols && col < BCols) { // A is transposed, use ACols for row checks
        float sum = 0.0f;
        for (int e = 0; e < ARows; ++e) { // ARows is used here, reflecting the transposed dimension
            sum += A[e * ACols + row] * B[e * BCols + col]; // Access A as transposed
        }
        C[row * BCols + col] = sum;
    }
}

__global__ void ConvolutionKernel(
    const float* input, const float* kernel, float* output,
    size_t inputRows, size_t inputCols, size_t inputDepth,
    size_t kernelRows, size_t kernelCols, size_t kernelDepth,
    size_t outputRows, size_t outputCols,
    size_t stride, size_t padding)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < outputCols && y < outputRows)
    {
        float sum = 0.0f;
        for (int ky = 0; ky < kernelRows; ky++)
        {
            for (int kx = 0; kx < kernelCols; kx++)
            {
                for (int d = 0; d < kernelDepth; d++)
                {
                    int posY = y * stride + ky - padding;
                    int posX = x * stride + kx - padding;

                    if (posY >= 0 && posY < inputRows && posX >= 0 && posX < inputCols) {
                        int inputIndex = d * inputRows * inputCols + (posY * inputCols + posX);
                        int kernelIndex = d * kernelRows * kernelCols + (ky * kernelCols + kx);
                        sum += input[inputIndex] * kernel[kernelIndex];
                    }
                }
            }
        }
        int outputIndex = y * outputCols + x;
        output[outputIndex] += sum;
    }
}

__global__ void Convolution3DKernel(
    const float* input, const float* kernel, float* output,
    size_t inputRows, size_t inputCols, size_t inputDepth,
    size_t kernelRows, size_t kernelCols,
    size_t outputRows, size_t outputCols, size_t outputDepth,
    size_t stride, size_t padding)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int d = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < outputCols && y < outputRows && d < outputDepth)
    {
        float sum = 0.0f;
        for (int ky = 0; ky < kernelRows; ky++)
        {
            for (int kx = 0; kx < kernelCols; kx++)
            {
                int posY = y * stride + ky - padding;
                int posX = x * stride + kx - padding;

                if (posY >= 0 && posY < inputRows && posX >= 0 && posX < inputCols) {
                    int inputIndex = d * inputRows * inputCols + (posY * inputCols + posX);
                    int kernelIndex = ky * kernelCols + kx;
                    sum += input[inputIndex] * kernel[kernelIndex];
                }
            }
        }
        int outputIndex = d * outputRows * outputCols + (y * outputCols + x);
        output[outputIndex] += sum;
    }
}

__global__ void Convolution3DBackKernelFlipKernel(
    const float* input, const float* kernel, float* output,
    size_t inputRows, size_t inputCols,
    size_t kernelRows, size_t kernelCols, size_t kernelDepth,
    size_t outputRows, size_t outputCols, size_t outputDepth,
    size_t stride, size_t padding)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int d = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < outputCols && y < outputRows && d < outputDepth)
    {
        float sum = 0.0f;
        for (int ky = 0; ky < kernelRows; ky++)
        {
            for (int kx = 0; kx < kernelCols; kx++)
            {
                int posY = y * stride + ky - padding;
                int posX = x * stride + kx - padding;

                size_t flippedKy = kernelRows - 1 - ky;
                size_t flippedKx = kernelCols - 1 - kx;

                if (posY >= 0 && posY < inputRows && posX >= 0 && posX < inputCols) {
                    int inputIndex = posY * inputCols + posX;
                    int kernelIndex = d * kernelRows * kernelCols + (ky * kernelCols + kx);
                    sum += input[inputIndex] * kernel[kernelIndex];
                }
            }
        }
        int outputIndex = d * outputRows * outputCols + (y * outputCols + x);
        output[outputIndex] += sum;
    }
}

__global__ void MaxPoolKernel(
    const float* input, float* output,
    size_t inputRows, size_t inputCols, size_t inputDepth,
    size_t outputRows, size_t outputCols, size_t outputDepth,
    size_t poolHeight, size_t poolWidth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int d = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < outputCols && y < outputRows && d < outputDepth) {
        float maxVal = -FLT_MAX;
        for (int py = 0; py < poolHeight; py++)
        {
            for (int px = 0; px < poolWidth; px++)
            {
                int inputY = y * poolHeight + py;
                int inputX = x * poolWidth + px;

                if (inputY < inputRows && inputX < inputCols) {
                    int inputIndex = d * inputRows * inputCols + (inputY * inputCols + inputX);
                    float val = input[inputIndex];
                    if (val > maxVal)
                    {
                        maxVal = val;
                    }
                }
            }
        }
        int outputIndex = d * outputRows * outputCols + (y * outputCols + x);
        output[outputIndex] = maxVal;
    }
}

__global__ void DistributedReverseMaxPoolKernel(
    const float* input, const float* output, float* distributed,
    size_t inputRows, size_t inputCols, size_t inputDepth,
    size_t outputRows, size_t outputCols, size_t outputDepth,
    size_t distributedRows, size_t distributedCols, size_t distributedDepth,
    size_t poolHeight, size_t poolWidth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int d = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < outputCols && y < outputRows && d < outputDepth) {
        float maxVal = -FLT_MAX;
        int poolX = 0, poolY = 0;
        for (int py = 0; py < poolHeight; py++)
        {
            for (int px = 0; px < poolWidth; px++)
            {
                int inputY = y * poolHeight + py;
                int inputX = x * poolWidth + px;

                if (inputY < inputRows && inputX < inputCols) {
                    int inputIndex = d * inputRows * inputCols + (inputY * inputCols + inputX);
                    float val = input[inputIndex];
                    if (val > maxVal)
                    {
                        maxVal = val;
                        poolX = px;
                        poolY = py;
                    }
                }
            }
        }
        int distributeY = y * poolHeight + poolY;
        int distributeX = x * poolWidth + poolX;
        int outputIndex = d * outputRows * outputCols + (y * outputCols + x);
        int distributedIndex = d * distributedRows * distributedCols + (distributeY * distributedCols + distributeX);
        distributed[distributedIndex] = output[outputIndex];
    }
}

__global__ void NearestUpsampleKernel(
    const float* input, float* output,
    size_t inputRows, size_t inputCols, size_t inputDepth,
    size_t outputRows, size_t outputCols, size_t outputDepth,
    size_t upsampleHeight, size_t upsampleWidth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int d = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < outputCols && y < outputRows && d < outputDepth) {
        int inputY = y / upsampleHeight;
        int inputX = x / upsampleWidth;
        int inputD = d;

        int outputIndex = (y * outputCols + x) + d * outputCols * outputRows;
        int inputIndex = (inputY * inputCols + inputX) + d * inputCols * inputRows;

        output[outputIndex] = input[inputIndex];
    }
}

__global__ void DistributeReverseNearestUpsampleKernel(
    const float* output, float* distributed,
    size_t outputRows, size_t outputCols, size_t outputDepth,
    size_t distributedRows, size_t distributedCols, size_t distributedDepth,
    size_t upsampleHeight, size_t upsampleWidth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int d = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < distributedCols && y < distributedRows && d < distributedDepth) {
        float value = 0.0f;
        for (int uy = 0; uy < upsampleHeight; uy++)
        {
            for (int ux = 0; ux < upsampleWidth; ux++)
            {
                int outputY = y * upsampleHeight + uy;
                int outputX = x * upsampleWidth + ux;

                int outputIndex = (outputY * outputCols + outputX) + d * outputRows * outputCols;
                value += output[outputIndex];
            }
        }
        int distributedIndex = (y * distributedCols + x) + d * distributedRows * distributedCols;
        distributed[distributedIndex] = value;
    }
}

__global__ void DropOutKernel(const float* input, float* output, size_t size, float dropoutRate, float retentionProb, float* dropOutMask)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        hiprandState state;
        hiprand_init(1234, idx, 0, &state);

        float r = hiprand_uniform(&state);
        if (r > dropoutRate)
        {
            output[idx] = input[idx] / retentionProb;
            if (dropOutMask)
            {
                dropOutMask[idx] = 1.0f;
            }
        }
    }
}

__global__ void RandomizeKernel(float* output, size_t size, float min, float max)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        hiprandState state;
        hiprand_init(1234, idx, 0, &state);
        float r = hiprand_uniform(&state);
        output[idx] = min + (max - min) * r;
    }
}

__global__ void CrossEntropyLossKernel(const float* targets, const float* predictions, float* loss, size_t size)
{
    const float epsilon = 0.00000001f;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        if(predictions[idx] > epsilon)
        {
            loss[idx] = (targets[idx] / predictions[idx]) / (float)size;
        }
        else
        {
            loss[idx] = 0;
        }
    }
}